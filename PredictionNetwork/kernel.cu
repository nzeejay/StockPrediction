
#include <hip/hip_runtime.h>
﻿//Kernel code:
extern "C"
{
	__device__ int getIndex(int x, int y, int z, int X, int Y) {
		return (x + (y * Y)) + (z * X * Y);
	}

	__device__ float sigmoid(float f) {
		return 1 / (1 + __expf(-f));
	}

	__device__ float sigmoidDer(float f) {
		return f * (1 - f);
	}

	//activations
	__global__ void Sigmoid(float*bpData, float* data) {
		int ID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);

		data[ID] = sigmoid(bpData[ID]);
	}

	// Device code
	__global__ void Forward(float* data, float* weights, float* prev) {

		//__shared__ float thisNode;
		//
		//__syncthreads();

		int thisLayerID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);

		int prevLayerID = getIndex(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y);

		int weightID = thisLayerID * (blockDim.x * blockDim.y * blockDim.z) + prevLayerID;

		float val = prev[prevLayerID] * weights[weightID];

		//printf("%f \r\n", val);

		atomicAdd(&data[thisLayerID], val);

		//__syncthreads();
		//
		////activation function
		//if (threadIdx.x + threadIdx.y + threadIdx.z == 0) 
		//	data[thisLayerID] = sigmoid(thisNode);

	}

	__global__ void Backprop(float* data, float* bpData, float* weights, float* prev, float* error, float* prevError, float step) {
		int blockSize = (blockDim.x * blockDim.y * blockDim.z);

		int thisLayerID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);

		int prevLayerID = getIndex(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y);

		int weightID = thisLayerID * blockSize + prevLayerID;

		float prevActGD = weights[weightID] * sigmoidDer(bpData[prevLayerID]) * error[thisLayerID];
		atomicAdd(&prevError[prevLayerID], prevActGD);

		float weightGD = prev[prevLayerID] * sigmoidDer(bpData[prevLayerID]) * error[thisLayerID];
		weights[weightID] += -step * weightGD;
		
		//printf("%i %i %i %f %f\r\n", thisLayerID, prevLayerID, weightID, weightGD, prevActGD);
	}

	__global__ void Clear(float* data, float* bpData, float* error) {
		int ID = getIndex(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y);
		
		data[ID] = 0.f;
		bpData[ID] = 0.f;
		error[ID] = 0.f;
	}
}